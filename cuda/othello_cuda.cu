#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Constant board size
#define BOARD_SIZE 8
#define EMPTY 0
#define WHITE 1
#define BLACK 2

// Number of threads per block (can be tuned)
#define BLOCK_SIZE 256

// Coefficient structure for evaluation
typedef struct
{
  int material_coeff[3];
  int mobility_coeff[3];
  int corners_coeff[3];
  int parity_coeff[3];
  int stability_coeff[3];
  int frontier_coeff[3];
} EvaluationCoefficients;

// Game state structure
typedef struct
{
  int board[BOARD_SIZE][BOARD_SIZE];
  int player_color;
} GameState;

// Copy coefficients to device constant memory
__constant__ EvaluationCoefficients d_coeffs;

// CUDA kernel to evaluate multiple game states in parallel
__global__ void evaluateStatesKernel(GameState *states, int *scores, int num_states)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < num_states)
  {
    GameState state = states[idx];
    int board[BOARD_SIZE][BOARD_SIZE];
    int player_color = state.player_color;
    int opponent_color = (player_color == WHITE) ? BLACK : WHITE;

    // Copy board to local memory for faster access
    for (int i = 0; i < BOARD_SIZE; i++)
    {
      for (int j = 0; j < BOARD_SIZE; j++)
      {
        board[i][j] = state.board[i][j];
      }
    }

    // Count pieces for phase determination
    int piece_count = 0;
    for (int i = 0; i < BOARD_SIZE; i++)
    {
      for (int j = 0; j < BOARD_SIZE; j++)
      {
        if (board[i][j] != EMPTY)
        {
          piece_count++;
        }
      }
    }

    // Determine game phase (0=early, 1=mid, 2=late)
    int phase;
    if (piece_count < 20)
    {
      phase = 0;
    }
    else if (piece_count <= 58)
    {
      phase = 1;
    }
    else
    {
      phase = 2;
    }

    // Material evaluation
    int player_pieces = 0;
    int opponent_pieces = 0;
    for (int i = 0; i < BOARD_SIZE; i++)
    {
      for (int j = 0; j < BOARD_SIZE; j++)
      {
        if (board[i][j] == player_color)
        {
          player_pieces++;
        }
        else if (board[i][j] == opponent_color)
        {
          opponent_pieces++;
        }
      }
    }
    int material_score = player_pieces - opponent_pieces;

    // Corner evaluation
    int player_corners = 0;
    int opponent_corners = 0;
    if (board[0][0] == player_color)
      player_corners++;
    if (board[0][7] == player_color)
      player_corners++;
    if (board[7][0] == player_color)
      player_corners++;
    if (board[7][7] == player_color)
      player_corners++;

    if (board[0][0] == opponent_color)
      opponent_corners++;
    if (board[0][7] == opponent_color)
      opponent_corners++;
    if (board[7][0] == opponent_color)
      opponent_corners++;
    if (board[7][7] == opponent_color)
      opponent_corners++;

    int corner_score;
    if (player_corners + opponent_corners == 0)
    {
      corner_score = 0;
    }
    else
    {
      corner_score = 100 * (player_corners - opponent_corners) / (player_corners + opponent_corners);
    }

    // Simplified evaluation - using only material and corner for GPU implementation
    int final_score = d_coeffs.material_coeff[phase] * material_score +
                      d_coeffs.corners_coeff[phase] * corner_score;

    scores[idx] = final_score;
  }
}

// C wrapper functions for Go
extern "C"
{

  // Initialize CUDA and return success status
  __declspec(dllexport) int initCUDA()
  {
    hipError_t error;
    int deviceCount;

    error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess)
    {
      printf("CUDA Error: %s\n", hipGetErrorString(error));
      return 0;
    }

    if (deviceCount == 0)
    {
      printf("No CUDA-capable devices found\n");
      return 0;
    }

    // Choose device 0 by default
    error = hipSetDevice(0);
    if (error != hipSuccess)
    {
      printf("CUDA Error: %s\n", hipGetErrorString(error));
      return 0;
    }

    return 1;
  }

  // Set evaluation coefficients
  __declspec(dllexport) void setCoefficients(int *material, int *mobility, int *corners,
                                             int *parity, int *stability, int *frontier)
  {
    EvaluationCoefficients h_coeffs;

    // Copy coefficients from host arrays to host struct
    for (int i = 0; i < 3; i++)
    {
      h_coeffs.material_coeff[i] = material[i];
      h_coeffs.mobility_coeff[i] = mobility[i];
      h_coeffs.corners_coeff[i] = corners[i];
      h_coeffs.parity_coeff[i] = parity[i];
      h_coeffs.stability_coeff[i] = stability[i];
      h_coeffs.frontier_coeff[i] = frontier[i];
    }

    // Copy coefficients to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_coeffs), &h_coeffs, sizeof(EvaluationCoefficients));
  }

  // Evaluate multiple game states in parallel
  __declspec(dllexport) void evaluateStates(int *boards, int *player_colors, int *scores, int num_states)
  {
    GameState *h_states = (GameState *)malloc(num_states * sizeof(GameState));
    GameState *d_states;
    int *d_scores;

    // Prepare game states
    for (int s = 0; s < num_states; s++)
    {
      h_states[s].player_color = player_colors[s];
      for (int i = 0; i < BOARD_SIZE; i++)
      {
        for (int j = 0; j < BOARD_SIZE; j++)
        {
          h_states[s].board[i][j] = boards[s * BOARD_SIZE * BOARD_SIZE + i * BOARD_SIZE + j];
        }
      }
    }

    // Allocate device memory
    hipMalloc((void **)&d_states, num_states * sizeof(GameState));
    hipMalloc((void **)&d_scores, num_states * sizeof(int));

    // Copy data to device
    hipMemcpy(d_states, h_states, num_states * sizeof(GameState), hipMemcpyHostToDevice);

    // Calculate grid dimensions
    int blocks = (num_states + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Launch kernel
    evaluateStatesKernel<<<blocks, BLOCK_SIZE>>>(d_states, d_scores, num_states);

    // Copy results back to host
    hipMemcpy(scores, d_scores, num_states * sizeof(int), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_states);
    hipFree(d_scores);
    free(h_states);
  }

  // Free CUDA resources
  __declspec(dllexport) void cleanupCUDA()
  {
    hipDeviceReset();
  }

} // extern "C"
