#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "othello_cuda.h"

// Constant board size
#define BOARD_SIZE 8
#define EMPTY 0
#define WHITE 1
#define BLACK 2

// Increase BLOCK_SIZE for better GPU utilization
#define BLOCK_SIZE 256

// Maximum number of positions to store in transposition table
#define MAX_POSITIONS_POOL 65536
#define POSITIONS_BATCH_SIZE 1024

// Game state structure
typedef struct
{
  int board[BOARD_SIZE][BOARD_SIZE];
  int player_color;
} GameState;

// Global device variables
__constant__ EvaluationCoefficients d_coeffs;

// Host-side copies
EvaluationCoefficients h_coeffs;
hipError_t cuda_status = hipSuccess;

// Debug mode flag for more verbose output
#define DEBUG_MODE 0

// Add debug flag for printing evaluation details
#define DEBUG_EVAL 1

// Global coefficient arrays on device
__device__ int d_material_coeffs[3];
__device__ int d_mobility_coeffs[3];
__device__ int d_corners_coeffs[3];
__device__ int d_parity_coeffs[3];
__device__ int d_stability_coeffs[3];
__device__ int d_frontier_coeffs[3];

//-----------------------------------------------------------------------
// Device-only functions (run on GPU)
//-----------------------------------------------------------------------

// Check if a move is valid for the given board and player
__device__ bool isValidMove(int board[BOARD_SIZE][BOARD_SIZE], int player, int row, int col)
{
  // Check if the position is empty
  if (board[row][col] != EMPTY)
    return false;

  // Get opponent color
  int opponent = (player == WHITE) ? BLACK : WHITE;

  // Direction vectors for all 8 directions
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  // Check all 8 directions
  for (int dir = 0; dir < 8; dir++)
  {
    int r = row + dx[dir];
    int c = col + dy[dir];

    // First step must have opponent piece
    if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE && board[r][c] == opponent)
    {
      // Continue in this direction
      r += dx[dir];
      c += dy[dir];

      // Keep going until we find an empty cell, edge, or our own piece
      while (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE)
      {
        if (board[r][c] == EMPTY)
          break;
        if (board[r][c] == player)
          return true; // Found our own piece, move is valid

        // Continue in this direction
        r += dx[dir];
        c += dy[dir];
      }
    }
  }

  return false;
}

// Apply a move to the board and return a new board
__device__ void applyMove(int original[BOARD_SIZE][BOARD_SIZE], int result[BOARD_SIZE][BOARD_SIZE],
                          int player, int row, int col)
{
  // Copy the original board
  for (int i = 0; i < BOARD_SIZE; i++)
    for (int j = 0; j < BOARD_SIZE; j++)
      result[i][j] = original[i][j];

  // Place the piece
  result[row][col] = player;

  // Get opponent color
  int opponent = (player == WHITE) ? BLACK : WHITE;

  // Direction vectors for all 8 directions
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  // Check all 8 directions and flip pieces
  for (int dir = 0; dir < 8; dir++)
  {
    int r = row + dx[dir];
    int c = col + dy[dir];

    // Pieces to flip in this direction
    int flip_r[BOARD_SIZE * BOARD_SIZE], flip_c[BOARD_SIZE * BOARD_SIZE];
    int flip_count = 0;

    // Check if first piece is opponent
    if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE && result[r][c] == opponent)
    {
      // Remember this piece
      flip_r[flip_count] = r;
      flip_c[flip_count] = c;
      flip_count++;

      // Continue in this direction
      r += dx[dir];
      c += dy[dir];

      // Find all opponent pieces in this direction
      while (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE)
      {
        if (result[r][c] == EMPTY)
        {
          flip_count = 0; // No pieces to flip
          break;
        }

        if (result[r][c] == player)
          break; // Found our piece, can flip

        // Remember opponent piece
        flip_r[flip_count] = r;
        flip_c[flip_count] = c;
        flip_count++;

        // Continue in this direction
        r += dx[dir];
        c += dy[dir];
      }

      // If we found our piece at the end, flip all pieces in between
      if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE && result[r][c] == player)
      {
        for (int i = 0; i < flip_count; i++)
          result[flip_r[i]][flip_c[i]] = player;
      }
    }
  }
}

// Get all valid moves for a player
__device__ int getValidMoves(int board[BOARD_SIZE][BOARD_SIZE], int player, int moves_r[64], int moves_c[64])
{
  int count = 0;
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (isValidMove(board, player, i, j))
      {
        moves_r[count] = i;
        moves_c[count] = j;
        count++;
      }
    }
  }
  return count;
}

// Evaluate stability of pieces
__device__ int evaluateStability(int board[BOARD_SIZE][BOARD_SIZE], int player, int opponent)
{
  // Pre-computed stability weights
  const int stability_map[BOARD_SIZE][BOARD_SIZE] = {
      {4, -3, 2, 2, 2, 2, -3, 4},
      {-3, -4, -1, -1, -1, -1, -4, -3},
      {2, -1, 1, 0, 0, 1, -1, 2},
      {2, -1, 0, 1, 1, 0, -1, 2},
      {2, -1, 0, 1, 1, 0, -1, 2},
      {2, -1, 1, 0, 0, 1, -1, 2},
      {-3, -4, -1, -1, -1, -1, -4, -3},
      {4, -3, 2, 2, 2, 2, -3, 4}};

  int player_stability = 0;
  int opponent_stability = 0;

  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board[i][j] == player)
      {
        player_stability += stability_map[i][j];
      }
      else if (board[i][j] == opponent)
      {
        opponent_stability += stability_map[i][j];
      }
    }
  }

  return player_stability - opponent_stability;
}

// Count frontier discs (adjacent to empty spaces)
__device__ int countFrontierDiscs(int board[BOARD_SIZE][BOARD_SIZE], int player)
{
  int count = 0;
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board[i][j] == player)
      {
        // Check if this piece is adjacent to an empty square
        for (int dir = 0; dir < 8; dir++)
        {
          int r = i + dx[dir];
          int c = j + dy[dir];

          if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE &&
              board[r][c] == EMPTY)
          {
            count++;
            break; // Count each piece only once
          }
        }
      }
    }
  }

  return count;
}

// Enhanced board evaluation with more heuristics - made completely deterministic
__device__ int evaluateBoard(int *board, int player, int phase)
{
  // Convert the flat board array to 2D array
  int board_2d[BOARD_SIZE][BOARD_SIZE];
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      board_2d[i][j] = board[i * BOARD_SIZE + j];
    }
  }

  int opponent = (player == WHITE) ? BLACK : WHITE;

  // Count pieces for phase
  int piece_count = 0;
  int player_pieces = 0;
  int opponent_pieces = 0;

  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board_2d[i][j] != EMPTY)
      {
        piece_count++;
        if (board_2d[i][j] == player)
          player_pieces++;
        else if (board_2d[i][j] == opponent)
          opponent_pieces++;
      }
    }
  }

  // Get all raw score components and coefficients
  // material
  int material_score = player_pieces - opponent_pieces;

  // corners
  int player_corners = 0;
  int opponent_corners = 0;
  if (board_2d[0][0] == player)
    player_corners++;
  if (board_2d[0][7] == player)
    player_corners++;
  if (board_2d[7][0] == player)
    player_corners++;
  if (board_2d[7][7] == player)
    player_corners++;
  if (board_2d[0][0] == opponent)
    opponent_corners++;
  if (board_2d[0][7] == opponent)
    opponent_corners++;
  if (board_2d[7][0] == opponent)
    opponent_corners++;
  if (board_2d[7][7] == opponent)
    opponent_corners++;
  int corner_score = player_corners - opponent_corners;

  // mobility
  int moves_r[64], moves_c[64];
  int player_moves = getValidMoves(board_2d, player, moves_r, moves_c);
  int opponent_moves = getValidMoves(board_2d, opponent, moves_r, moves_c);
  int mobility_score = player_moves - opponent_moves;

  // parity
  int empty_squares = 64 - player_pieces - opponent_pieces;
  int parity_score = 0;
  if (empty_squares % 2 == 0)
  {
    parity_score = (player == BLACK) ? -1 : 1;
  }
  else
  {
    parity_score = (player == BLACK) ? 1 : -1;
  }

  // stability calculation
  const int stability_map[BOARD_SIZE][BOARD_SIZE] = {
      {4, -3, 2, 2, 2, 2, -3, 4},
      {-3, -4, -1, -1, -1, -1, -4, -3},
      {2, -1, 1, 0, 0, 1, -1, 2},
      {2, -1, 0, 1, 1, 0, -1, 2},
      {2, -1, 0, 1, 1, 0, -1, 2},
      {2, -1, 1, 0, 0, 1, -1, 2},
      {-3, -4, -1, -1, -1, -1, -4, -3},
      {4, -3, 2, 2, 2, 2, -3, 4}};

  int player_stability = 0;
  int opponent_stability = 0;
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board_2d[i][j] == player)
      {
        player_stability += stability_map[i][j];
      }
      else if (board_2d[i][j] == opponent)
      {
        opponent_stability += stability_map[i][j];
      }
    }
  }
  int stability_score = player_stability - opponent_stability;

  // frontier
  int player_frontier = 0;
  int opponent_frontier = 0;
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board_2d[i][j] == player)
      {
        // Check if adjacent to empty
        for (int dir = 0; dir < 8; dir++)
        {
          int r = i + dx[dir];
          int c = j + dy[dir];
          if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE &&
              board_2d[r][c] == EMPTY)
          {
            player_frontier++;
            break;
          }
        }
      }
      else if (board_2d[i][j] == opponent)
      {
        // Check if adjacent to empty
        for (int dir = 0; dir < 8; dir++)
        {
          int r = i + dx[dir];
          int c = j + dy[dir];
          if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE &&
              board_2d[r][c] == EMPTY)
          {
            opponent_frontier++;
            break;
          }
        }
      }
    }
  }
  int frontier_score = opponent_frontier - player_frontier;

  // Calculate weighted components
  int material_contrib = d_coeffs.material_coeff[phase] * material_score;
  int mobility_contrib = d_coeffs.mobility_coeff[phase] * mobility_score;
  int corner_contrib = d_coeffs.corners_coeff[phase] * corner_score;
  int parity_contrib = d_coeffs.parity_coeff[phase] * parity_score;
  int stability_contrib = d_coeffs.stability_coeff[phase] * stability_score;
  int frontier_contrib = d_coeffs.frontier_coeff[phase] * frontier_score;

  // Final score
  int final_score = material_contrib + mobility_contrib + corner_contrib +
                    parity_contrib + stability_contrib + frontier_contrib;

  // Add detailed debug output if enabled
  if (DEBUG_EVAL)
  {
    printf("[GPU] P%d Phase=%d: Mat(%d*%d=%d) Mob(%d*%d=%d) Cor(%d*%d=%d) Par(%d*%d=%d) Stb(%d*%d=%d) Frt(%d*%d=%d) = %d\n",
           player, phase,
           d_coeffs.material_coeff[phase], material_score, material_contrib,
           d_coeffs.mobility_coeff[phase], mobility_score, mobility_contrib,
           d_coeffs.corners_coeff[phase], corner_score, corner_contrib,
           d_coeffs.parity_coeff[phase], parity_score, parity_contrib,
           d_coeffs.stability_coeff[phase], stability_score, stability_contrib,
           d_coeffs.frontier_coeff[phase], frontier_score, frontier_contrib,
           final_score);
  }

  return final_score;
}

// CUDA kernel to evaluate multiple game states in parallel with shared memory
__global__ void evaluateStatesKernel(GameState *states, int *scores, int num_states)
{
  // Use shared memory for faster access
  __shared__ int shared_results[BLOCK_SIZE];

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;

  if (idx < num_states)
  {
    GameState state = states[idx];
    int board[BOARD_SIZE][BOARD_SIZE];
    int player_color = state.player_color;

    // Copy the board to local memory for faster access
    for (int i = 0; i < BOARD_SIZE; i++)
    {
      for (int j = 0; j < BOARD_SIZE; j++)
      {
        board[i][j] = state.board[i][j];
      }
    }

    // Flatten the 2D board into a 1D array
    int flatBoard[64];
    for (int i = 0; i < BOARD_SIZE; i++)
    {
      for (int j = 0; j < BOARD_SIZE; j++)
      {
        flatBoard[i * BOARD_SIZE + j] = board[i][j];
      }
    }

    // Determine game phase
    int piece_count = 0;
    for (int i = 0; i < BOARD_SIZE; i++)
    {
      for (int j = 0; j < BOARD_SIZE; j++)
      {
        if (board[i][j] != EMPTY)
        {
          piece_count++;
        }
      }
    }

    int phase;
    if (piece_count < 20)
      phase = 0;
    else if (piece_count <= 58)
      phase = 1;
    else
      phase = 2;

    // Calculate and store the evaluation score
    // Note: evaluateBoard returns score with opposite sign to CPU version
    shared_results[tid] = evaluateBoard(flatBoard, player_color, phase);
  }
  else
  {
    // Default value for unused threads
    shared_results[tid] = 0;
  }

  // Synchronize threads in the block
  __syncthreads();

  // Copy result to global memory
  if (idx < num_states)
  {
    scores[idx] = shared_results[tid];
  }
}

//-----------------------------------------------------------------------
// Host-only code (CPU side)
//-----------------------------------------------------------------------

// Host function to check if a move is valid (CPU implementation)
bool isValidMoveHost(int board[BOARD_SIZE][BOARD_SIZE], int player, int row, int col)
{
  if (board[row][col] != EMPTY)
    return false;

  int opponent = (player == WHITE) ? BLACK : WHITE;
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  for (int dir = 0; dir < 8; dir++)
  {
    int r = row + dx[dir];
    int c = col + dy[dir];

    if (r < 0 || r >= BOARD_SIZE || c < 0 || c >= BOARD_SIZE || board[r][c] != opponent)
      continue;

    r += dx[dir];
    c += dy[dir];
    bool foundPlayerPiece = false;

    while (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE)
    {
      if (board[r][c] == EMPTY)
        break;
      if (board[r][c] == player)
      {
        foundPlayerPiece = true;
        break;
      }
      r += dx[dir];
      c += dy[dir];
    }

    if (foundPlayerPiece)
      return true;
  }

  return false;
}

// Host function to apply a move (CPU implementation)
void applyMoveHost(int board[BOARD_SIZE][BOARD_SIZE], int player, int row, int col)
{
  board[row][col] = player;
  int opponent = (player == WHITE) ? BLACK : WHITE;
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  for (int dir = 0; dir < 8; dir++)
  {
    int r = row + dx[dir];
    int c = col + dy[dir];

    if (r < 0 || r >= BOARD_SIZE || c < 0 || c >= BOARD_SIZE || board[r][c] != opponent)
      continue;

    // Store positions to flip
    int flipPositions[BOARD_SIZE * BOARD_SIZE][2];
    int flipCount = 0;

    while (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE && board[r][c] == opponent)
    {
      flipPositions[flipCount][0] = r;
      flipPositions[flipCount][1] = c;
      flipCount++;
      r += dx[dir];
      c += dy[dir];
    }

    // If we found our piece at the end, flip all pieces in between
    if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE && board[r][c] == player)
    {
      for (int i = 0; i < flipCount; i++)
      {
        board[flipPositions[i][0]][flipPositions[i][1]] = player;
      }
    }
  }
}

// Host function to get all valid moves (CPU implementation)
int getValidMovesHost(int board[BOARD_SIZE][BOARD_SIZE], int player, int moves_r[64], int moves_c[64])
{
  int count = 0;
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (isValidMoveHost(board, player, i, j))
      {
        moves_r[count] = i;
        moves_c[count] = j;
        count++;
      }
    }
  }
  return count;
}

// Evaluate a board state (CPU implementation)
int evaluateBoardHost(int board[BOARD_SIZE][BOARD_SIZE], int player, EvaluationCoefficients coeffs)
{
  int opponent = (player == WHITE) ? BLACK : WHITE;

  // Count pieces for phase determination
  int piece_count = 0;
  int player_pieces = 0;
  int opponent_pieces = 0;

  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board[i][j] != EMPTY)
      {
        piece_count++;
        if (board[i][j] == player)
          player_pieces++;
        else if (board[i][j] == opponent)
          opponent_pieces++;
      }
    }
  }

  int phase;
  if (piece_count < 20)
    phase = 0; // Early game
  else if (piece_count <= 58)
    phase = 1; // Mid game
  else
    phase = 2; // Late game

  // Material evaluation
  int material_score = player_pieces - opponent_pieces;

  // Corner evaluation
  int player_corners = 0;
  int opponent_corners = 0;
  if (board[0][0] == player)
    player_corners++;
  if (board[0][7] == player)
    player_corners++;
  if (board[7][0] == player)
    player_corners++;
  if (board[7][7] == player)
    player_corners++;

  if (board[0][0] == opponent)
    opponent_corners++;
  if (board[0][7] == opponent)
    opponent_corners++;
  if (board[7][0] == opponent)
    opponent_corners++;
  if (board[7][7] == opponent)
    opponent_corners++;

  // Simple difference for corner score - match device/Go implementation
  int corner_score = player_corners - opponent_corners;

  // Mobility calculation
  int moves_r[64], moves_c[64];
  int player_moves = getValidMovesHost(board, player, moves_r, moves_c);
  int opponent_moves = getValidMovesHost(board, opponent, moves_r, moves_c);

  // Simple difference for mobility - match device/Go implementation
  int mobility_score = player_moves - opponent_moves;

  // Parity evaluation - exactly match device implementation
  int empty_squares = 64 - player_pieces - opponent_pieces;
  int parity_score = 0;
  if (empty_squares % 2 == 0)
  {
    parity_score = (player == BLACK) ? -1 : 1;
  }
  else
  {
    parity_score = (player == BLACK) ? 1 : -1;
  }

  // Stability evaluation using the same map
  const int stability_map[BOARD_SIZE][BOARD_SIZE] = {
      {4, -3, 2, 2, 2, 2, -3, 4},
      {-3, -4, -1, -1, -1, -1, -4, -3},
      {2, -1, 1, 0, 0, 1, -1, 2},
      {2, -1, 0, 1, 1, 0, -1, 2},
      {2, -1, 0, 1, 1, 0, -1, 2},
      {2, -1, 1, 0, 0, 1, -1, 2},
      {-3, -4, -1, -1, -1, -1, -4, -3},
      {4, -3, 2, 2, 2, 2, -3, 4}};

  int player_stability = 0;
  int opponent_stability = 0;
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board[i][j] == player)
      {
        player_stability += stability_map[i][j];
      }
      else if (board[i][j] == opponent)
      {
        opponent_stability += stability_map[i][j];
      }
    }
  }
  int stability_score = player_stability - opponent_stability;

  // Frontier discs calculation - must match device implementation exactly
  int player_frontier = 0;
  int opponent_frontier = 0;
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board[i][j] == player)
      {
        // Check if this piece is adjacent to an empty square
        for (int dir = 0; dir < 8; dir++)
        {
          int r = i + dx[dir];
          int c = j + dy[dir];

          if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE && board[r][c] == EMPTY)
          {
            player_frontier++;
            break; // Count each piece only once
          }
        }
      }
      else if (board[i][j] == opponent)
      {
        // Check if this piece is adjacent to an empty square
        for (int dir = 0; dir < 8; dir++)
        {
          int r = i + dx[dir];
          int c = j + dy[dir];

          if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE && board[r][c] == EMPTY)
          {
            opponent_frontier++;
            break; // Count each piece only once
          }
        }
      }
    }
  }

  // Simple difference for frontier score - match device/Go implementation
  int frontier_score = opponent_frontier - player_frontier;

  // Calculate each component's contribution
  int material_contrib = coeffs.material_coeff[phase] * material_score;
  int mobility_contrib = coeffs.mobility_coeff[phase] * mobility_score;
  int corner_contrib = coeffs.corners_coeff[phase] * corner_score;
  int parity_contrib = coeffs.parity_coeff[phase] * parity_score;
  int stability_contrib = coeffs.stability_coeff[phase] * stability_score;
  int frontier_contrib = coeffs.frontier_coeff[phase] * frontier_score;

  // Final weighted score
  int final_score = material_contrib + mobility_contrib + corner_contrib +
                    parity_contrib + stability_contrib + frontier_contrib;

  // Add detailed debug output matching the GPU version
  if (DEBUG_EVAL)
  {
    printf("[CPU] P%d Phase=%d: Mat(%d*%d=%d) Mob(%d*%d=%d) Cor(%d*%d=%d) Par(%d*%d=%d) Stb(%d*%d=%d) Frt(%d*%d=%d) = %d\n",
           player, phase,
           coeffs.material_coeff[phase], material_score, material_contrib,
           coeffs.mobility_coeff[phase], mobility_score, mobility_contrib,
           coeffs.corners_coeff[phase], corner_score, corner_contrib,
           coeffs.parity_coeff[phase], parity_score, parity_contrib,
           coeffs.stability_coeff[phase], stability_score, stability_contrib,
           coeffs.frontier_coeff[phase], frontier_score, frontier_contrib,
           final_score);
  }

  return final_score;
}

// Recursive minimax search (CPU implementation)
int minimaxHost(int board[BOARD_SIZE][BOARD_SIZE], int player, int depth, bool maximizing,
                int alpha, int beta, int *best_row, int *best_col, EvaluationCoefficients coeffs)
{
  // Leaf node evaluation
  if (depth == 0 || isGameFinishedHost(board))
  {
    return evaluateBoardHost(board, player, coeffs);
  }

  int opponent = (player == WHITE) ? BLACK : WHITE;
  int moves_r[64], moves_c[64];
  int move_count;

  // Determine whose turn it is
  int current_player = maximizing ? player : opponent;

  // Get valid moves for current player
  move_count = getValidMovesHost(board, current_player, moves_r, moves_c);

  // Sort moves to ensure consistent ordering with Go implementation
  for (int i = 0; i < move_count - 1; i++)
  {
    for (int j = i + 1; j < move_count; j++)
    {
      // Sort by row first, then column (same as Go implementation)
      if (moves_r[i] > moves_r[j] || (moves_r[i] == moves_r[j] && moves_c[i] > moves_c[j]))
      {
        // Swap rows
        int temp_r = moves_r[i];
        moves_r[i] = moves_r[j];
        moves_r[j] = temp_r;

        // Swap columns
        int temp_c = moves_c[i];
        moves_c[i] = moves_c[j];
        moves_c[j] = temp_c;
      }
    }
  }

  // No moves - check if game is over or we need to pass
  if (move_count == 0)
  {
    // Check if opponent has moves
    int opp_player = maximizing ? opponent : player;
    int opp_moves_r[64], opp_moves_c[64];
    int opp_move_count = getValidMovesHost(board, opp_player, opp_moves_r, opp_moves_c);

    // Game is over
    if (opp_move_count == 0)
    {
      return evaluateBoardHost(board, player, coeffs);
    }

    // Pass turn to other player - same depth but flip maximizing
    return minimaxHost(board, player, depth - 1, !maximizing, alpha, beta, best_row, best_col, coeffs);
  }

  // Maximizing player (our player)
  if (maximizing)
  {
    int best_score = -1000000;
    int best_move_r = -1;
    int best_move_c = -1;

    for (int i = 0; i < move_count; i++)
    {
      // Copy board
      int new_board[BOARD_SIZE][BOARD_SIZE];
      for (int r = 0; r < BOARD_SIZE; r++)
        for (int c = 0; c < BOARD_SIZE; c++)
          new_board[r][c] = board[r][c];

      // Apply move
      applyMoveHost(new_board, player, moves_r[i], moves_c[i]);

      // Recursive search
      int dummy_r = -1, dummy_c = -1;
      int score = minimaxHost(new_board, player, depth - 1, false, alpha, beta, &dummy_r, &dummy_c, coeffs);

      // Use strict '>' comparison for consistent selection with Go implementation
      if (score > best_score)
      {
        best_score = score;
        best_move_r = moves_r[i];
        best_move_c = moves_c[i];
      }

      // Alpha-beta pruning
      alpha = (alpha > best_score) ? alpha : best_score;
      if (beta <= alpha)
        break;
    }

    *best_row = best_move_r;
    *best_col = best_move_c;
    return best_score;
  }
  // Minimizing player (opponent)
  else
  {
    int best_score = 1000000;
    int best_move_r = -1;
    int best_move_c = -1;

    for (int i = 0; i < move_count; i++)
    {
      // Copy board
      int new_board[BOARD_SIZE][BOARD_SIZE];
      for (int r = 0; r < BOARD_SIZE; r++)
        for (int c = 0; c < BOARD_SIZE; c++)
          new_board[r][c] = board[r][c];

      // Apply move
      applyMoveHost(new_board, opponent, moves_r[i], moves_c[i]);

      // Recursive search
      int dummy_r = -1, dummy_c = -1;
      int score = minimaxHost(new_board, player, depth - 1, true, alpha, beta, &dummy_r, &dummy_c, coeffs);

      // Use strict '<' comparison for consistent selection with Go implementation
      if (score < best_score)
      {
        best_score = score;
        best_move_r = moves_r[i];
        best_move_c = moves_c[i];
      }

      // Alpha-beta pruning
      beta = (beta < best_score) ? beta : best_score;
      if (beta <= alpha)
        break;
    }

    *best_row = best_move_r;
    *best_col = best_move_c;
    return best_score;
  }
}

// Add a helper function to check if game is finished
bool isGameFinishedHost(int board[BOARD_SIZE][BOARD_SIZE])
{
  int black_moves_r[64], black_moves_c[64];
  int white_moves_r[64], white_moves_c[64];

  int black_move_count = getValidMovesHost(board, BLACK, black_moves_r, black_moves_c);
  int white_move_count = getValidMovesHost(board, WHITE, white_moves_r, white_moves_c);

  return black_move_count == 0 && white_move_count == 0;
}

//-----------------------------------------------------------------------
// External C interface (exported functions)
//-----------------------------------------------------------------------

// Initialize CUDA and return success status
__declspec(dllexport) int initCUDA()
{
  hipError_t error;
  int deviceCount;

  error = hipGetDeviceCount(&deviceCount);
  if (error != hipSuccess)
  {
    printf("CUDA Error: %s\n", hipGetErrorString(error));
    return 0;
  }

  if (deviceCount == 0)
  {
    printf("No CUDA-capable devices found\n");
    return 0;
  }

  // Choose device 0 by default
  error = hipSetDevice(0);
  if (error != hipSuccess)
  {
    printf("CUDA Error: %s\n", hipGetErrorString(error));
    return 0;
  }

  return 1;
}

// Evaluate multiple game states in parallel
__declspec(dllexport) void evaluateStates(int *boards, int *player_colors, int *scores, int num_states)
{
  // Measure transfer and execution time for profiling
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  GameState *h_states = (GameState *)malloc(num_states * sizeof(GameState));
  GameState *d_states;
  int *d_scores;

  // Prepare game states
  for (int s = 0; s < num_states; s++)
  {
    h_states[s].player_color = player_colors[s];
    for (int i = 0; i < BOARD_SIZE; i++)
    {
      for (int j = 0; j < BOARD_SIZE; j++)
      {
        h_states[s].board[i][j] = boards[s * BOARD_SIZE * BOARD_SIZE + i * BOARD_SIZE + j];
      }
    }
  }

  // Allocate device memory and copy data - use pinned memory for faster transfers
  hipMalloc((void **)&d_states, num_states * sizeof(GameState));
  hipMalloc((void **)&d_scores, num_states * sizeof(int));

  // Copy data to device
  hipMemcpy(d_states, h_states, num_states * sizeof(GameState), hipMemcpyHostToDevice);

  // Calculate grid dimensions
  int threads = BLOCK_SIZE;
  int blocks = (num_states + threads - 1) / threads;

  // Launch kernel
  evaluateStatesKernel<<<blocks, threads>>>(d_states, d_scores, num_states);

  // Check for kernel launch errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("CUDA Kernel Error: %s\n", hipGetErrorString(err));
    // In case of error, zero all scores
    memset(scores, 0, num_states * sizeof(int));

    // Free memory and return
    hipFree(d_states);
    hipFree(d_scores);
    free(h_states);
    return;
  }

  // Copy results back to host
  hipMemcpy(scores, d_scores, num_states * sizeof(int), hipMemcpyDeviceToHost);

  // Remove extra sign flip (previously, we had: for (int i = 0; i < num_states; i++) { scores[i] = -scores[i]; })

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  if (num_states > 1000)
  {
    printf("GPU processed %d states in %.2f ms (%.2f states/ms)\n",
           num_states, milliseconds, num_states / milliseconds);
  }

  // Free memory
  hipFree(d_states);
  hipFree(d_scores);
  free(h_states);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

// Find the best move using minimax
__declspec(dllexport) int findBestMove(int *board, int player_color, int depth, int *best_row, int *best_col)
{
  // Convert the flat board array to 2D array
  int board_2d[BOARD_SIZE][BOARD_SIZE];
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      board_2d[i][j] = board[i * BOARD_SIZE + j];
    }
  }

  // Get current evaluation coefficients from global memory
  EvaluationCoefficients h_coeffs;
  hipMemcpyFromSymbol(&h_coeffs, HIP_SYMBOL(d_coeffs), sizeof(EvaluationCoefficients));

  // Initialize best move coordinates
  int br = -1, bc = -1;

  // Get valid moves and check for special cases first
  int moves_r[64], moves_c[64];
  int move_count = getValidMovesHost(board_2d, player_color, moves_r, moves_c);

  // No valid moves
  if (move_count == 0)
  {
    *best_row = -1;
    *best_col = -1;
    return -1000000;
  }

  // If only one move, return it immediately
  if (move_count == 1)
  {
    *best_row = moves_r[0];
    *best_col = moves_c[0];

    // Apply the move to get a more accurate score
    int new_board[BOARD_SIZE][BOARD_SIZE];
    for (int i = 0; i < BOARD_SIZE; i++)
      for (int j = 0; j < BOARD_SIZE; j++)
        new_board[i][j] = board_2d[i][j];

    applyMoveHost(new_board, player_color, moves_r[0], moves_c[0]);

    // Get score from the position after our move
    return evaluateBoardHost(new_board, player_color, h_coeffs);
  }

  // Sort the moves for deterministic processing
  for (int i = 0; i < move_count - 1; i++)
  {
    for (int j = i + 1; j < move_count; j++)
    {
      if (moves_r[i] > moves_r[j] || (moves_r[i] == moves_r[j] && moves_c[i] > moves_c[j]))
      {
        // Swap rows
        int temp_r = moves_r[i];
        moves_r[i] = moves_r[j];
        moves_r[j] = temp_r;

        // Swap columns
        int temp_c = moves_c[i];
        moves_c[i] = moves_c[j];
        moves_c[j] = temp_c;
      }
    }
  }

  // Perform the minimax search with the sorted moves
  int best_score = minimaxHost(
      board_2d,     // game board
      player_color, // current player
      depth,        // search depth
      true,         // maximizing player
      -1000000,     // alpha
      1000000,      // beta
      &br,          // best row (output)
      &bc,          // best column (output)
      h_coeffs      // evaluation coefficients
  );

  // Set output parameters
  *best_row = br;
  *best_col = bc;

  // Return consistent score (no sign flip needed here since minimaxHost already uses CPU evaluation)
  return best_score;
}

// Check if a player has valid moves
__declspec(dllexport) int hasValidMoves(int *board, int player_color)
{
  // Convert the flat board array to 2D array
  int board_2d[BOARD_SIZE][BOARD_SIZE];
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      board_2d[i][j] = board[i * BOARD_SIZE + j];
    }
  }

  // Check for any valid move
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (isValidMoveHost(board_2d, player_color, i, j))
      {
        return 1; // At least one valid move exists
      }
    }
  }

  return 0; // No valid moves
}

// Check if game is finished (no valid moves for either player)
__declspec(dllexport) int isGameFinished(int *board)
{
  // No player can move = game is finished
  return !hasValidMoves(board, BLACK) && !hasValidMoves(board, WHITE);
}

// Get GPU memory information
__declspec(dllexport) void getGPUMemoryInfo(unsigned long long *free_memory, unsigned long long *total_memory)
{
  size_t free, total;
  hipMemGetInfo(&free, &total);

  *free_memory = free;
  *total_memory = total;
}

// Free CUDA resources
__declspec(dllexport) void cleanupCUDA()
{
  hipDeviceReset();
}

// Evaluate and find best moves for multiple positions in parallel
__declspec(dllexport) void evaluateAndFindBestMoves(int *boards, int *player_colors, int *depths,
                                                    int *scores, int *best_rows, int *best_cols, int num_states)
{
  // Measure transfer and execution time for profiling
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // Process each position sequentially
  // In a more optimized implementation, this would be done in parallel on the GPU
  for (int i = 0; i < num_states; i++)
  {
    // Extract the current board
    int *current_board = &boards[i * 64];
    int player_color = player_colors[i];
    int depth = depths[i];

    // Find best move for this position
    int row = -1, col = -1;
    int score = findBestMove(current_board, player_color, depth, &row, &col);

    // Store results
    scores[i] = score;
    best_rows[i] = row;
    best_cols[i] = col;
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  if (num_states > 10)
  {
    printf("GPU processed %d minimax positions in %.2f ms (%.2f positions/ms)\n",
           num_states, milliseconds, num_states / milliseconds);
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

// Set the evaluation coefficients on the GPU
extern "C" void setCoefficients(int *material, int *mobility, int *corners, int *parity, int *stability, int *frontier)
{
  hipError_t error;

  // Also set host-side coefficients for deterministic CPU evaluation
  memcpy(h_coeffs.material_coeff, material, 3 * sizeof(int));
  memcpy(h_coeffs.mobility_coeff, mobility, 3 * sizeof(int));
  memcpy(h_coeffs.corners_coeff, corners, 3 * sizeof(int));
  memcpy(h_coeffs.parity_coeff, parity, 3 * sizeof(int));
  memcpy(h_coeffs.stability_coeff, stability, 3 * sizeof(int));
  memcpy(h_coeffs.frontier_coeff, frontier, 3 * sizeof(int));

  // Use hipMemcpyToSymbol with explicit size and offset
  error = hipMemcpyToSymbol(HIP_SYMBOL(d_material_coeffs), material, 3 * sizeof(int), 0, hipMemcpyHostToDevice);
  if (error != hipSuccess)
  {
    printf("Error setting material coefficients: %s\n", hipGetErrorString(error));
    return;
  }

  error = hipMemcpyToSymbol(HIP_SYMBOL(d_mobility_coeffs), mobility, 3 * sizeof(int), 0, hipMemcpyHostToDevice);
  if (error != hipSuccess)
  {
    printf("Error setting mobility coefficients: %s\n", hipGetErrorString(error));
    return;
  }

  error = hipMemcpyToSymbol(HIP_SYMBOL(d_corners_coeffs), corners, 3 * sizeof(int), 0, hipMemcpyHostToDevice);
  if (error != hipSuccess)
  {
    printf("Error setting corners coefficients: %s\n", hipGetErrorString(error));
    return;
  }

  error = hipMemcpyToSymbol(HIP_SYMBOL(d_parity_coeffs), parity, 3 * sizeof(int), 0, hipMemcpyHostToDevice);
  if (error != hipSuccess)
  {
    printf("Error setting parity coefficients: %s\n", hipGetErrorString(error));
    return;
  }

  error = hipMemcpyToSymbol(HIP_SYMBOL(d_stability_coeffs), stability, 3 * sizeof(int), 0, hipMemcpyHostToDevice);
  if (error != hipSuccess)
  {
    printf("Error setting stability coefficients: %s\n", hipGetErrorString(error));
    return;
  }

  error = hipMemcpyToSymbol(HIP_SYMBOL(d_frontier_coeffs), frontier, 3 * sizeof(int), 0, hipMemcpyHostToDevice);
  if (error != hipSuccess)
  {
    printf("Error setting frontier coefficients: %s\n", hipGetErrorString(error));
    return;
  }

  // Ensure consistency by copying to coeffs structure for both CPU and GPU
  error = hipMemcpyToSymbol(HIP_SYMBOL(d_coeffs), &h_coeffs, sizeof(EvaluationCoefficients), 0, hipMemcpyHostToDevice);
  if (error != hipSuccess)
  {
    printf("Error setting full coefficients structure: %s\n", hipGetErrorString(error));
    return;
  }

  // Force synchronization to ensure all memory operations complete
  error = hipDeviceSynchronize();
  if (error != hipSuccess)
  {
    printf("Error in synchronization: %s\n", hipGetErrorString(error));
  }
}

// Add a debug function to export
__declspec(dllexport) int debugEvaluateBoard(int *board, int player_color, int *debug_info)
{
  // Convert the flat board array to 2D array
  int board_2d[BOARD_SIZE][BOARD_SIZE];
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      board_2d[i][j] = board[i * BOARD_SIZE + j];
    }
  }

  int opponent = (player_color == WHITE) ? BLACK : WHITE;

  // Count pieces for phase
  int piece_count = 0;
  int player_pieces = 0;
  int opponent_pieces = 0;

  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board_2d[i][j] != EMPTY)
      {
        piece_count++;
        if (board_2d[i][j] == player_color)
          player_pieces++;
        else if (board_2d[i][j] == opponent)
          opponent_pieces++;
      }
    }
  }

  int phase = piece_count < 20 ? 0 : (piece_count <= 58 ? 1 : 2);

  // Get current evaluation coefficients from global memory
  EvaluationCoefficients h_coeffs;
  hipMemcpyFromSymbol(&h_coeffs, HIP_SYMBOL(d_coeffs), sizeof(EvaluationCoefficients));

  // Get all raw score components and coefficients
  // material
  int material_score = player_pieces - opponent_pieces;

  // corners
  int player_corners = 0;
  int opponent_corners = 0;
  if (board_2d[0][0] == player_color)
    player_corners++;
  if (board_2d[0][7] == player_color)
    player_corners++;
  if (board_2d[7][0] == player_color)
    player_corners++;
  if (board_2d[7][7] == player_color)
    player_corners++;
  if (board_2d[0][0] == opponent)
    opponent_corners++;
  if (board_2d[0][7] == opponent)
    opponent_corners++;
  if (board_2d[7][0] == opponent)
    opponent_corners++;
  if (board_2d[7][7] == opponent)
    opponent_corners++;
  int corner_score = player_corners - opponent_corners;

  // mobility
  int moves_r[64], moves_c[64];
  int player_moves = getValidMovesHost(board_2d, player_color, moves_r, moves_c);
  int opponent_moves = getValidMovesHost(board_2d, opponent, moves_r, moves_c);
  int mobility_score = player_moves - opponent_moves;

  // parity
  int empty_squares = 64 - player_pieces - opponent_pieces;
  int parity_score = 0;
  if (empty_squares % 2 == 0)
  {
    parity_score = (player_color == BLACK) ? -1 : 1;
  }
  else
  {
    parity_score = (player_color == BLACK) ? 1 : -1;
  }

  // stability calculation
  const int stability_map[BOARD_SIZE][BOARD_SIZE] = {
      {4, -3, 2, 2, 2, 2, -3, 4},
      {-3, -4, -1, -1, -1, -1, -4, -3},
      {2, -1, 1, 0, 0, 1, -1, 2},
      {2, -1, 0, 1, 1, 0, -1, 2},
      {2, -1, 0, 1, 1, 0, -1, 2},
      {2, -1, 1, 0, 0, 1, -1, 2},
      {-3, -4, -1, -1, -1, -1, -4, -3},
      {4, -3, 2, 2, 2, 2, -3, 4}};

  int player_stability = 0;
  int opponent_stability = 0;
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board_2d[i][j] == player_color)
      {
        player_stability += stability_map[i][j];
      }
      else if (board_2d[i][j] == opponent)
      {
        opponent_stability += stability_map[i][j];
      }
    }
  }
  int stability_score = player_stability - opponent_stability;

  // frontier
  int player_frontier = 0;
  int opponent_frontier = 0;
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board_2d[i][j] == player_color)
      {
        // Check if adjacent to empty
        for (int dir = 0; dir < 8; dir++)
        {
          int r = i + dx[dir];
          int c = j + dy[dir];
          if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE &&
              board_2d[r][c] == EMPTY)
          {
            player_frontier++;
            break;
          }
        }
      }
      else if (board_2d[i][j] == opponent)
      {
        // Check if adjacent to empty
        for (int dir = 0; dir < 8; dir++)
        {
          int r = i + dx[dir];
          int c = j + dy[dir];
          if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE &&
              board_2d[r][c] == EMPTY)
          {
            opponent_frontier++;
            break;
          }
        }
      }
    }
  }
  int frontier_score = opponent_frontier - player_frontier;

  // Calculate weighted components
  int material_contrib = h_coeffs.material_coeff[phase] * material_score;
  int mobility_contrib = h_coeffs.mobility_coeff[phase] * mobility_score;
  int corner_contrib = h_coeffs.corners_coeff[phase] * corner_score;
  int parity_contrib = h_coeffs.parity_coeff[phase] * parity_score;
  int stability_contrib = h_coeffs.stability_coeff[phase] * stability_score;
  int frontier_contrib = h_coeffs.frontier_coeff[phase] * frontier_score;

  // Final score
  int final_score = material_contrib + mobility_contrib + corner_contrib +
                    parity_contrib + stability_contrib + frontier_contrib;

  // Show full breakdown if debug array provided
  if (debug_info != NULL)
  {
    // Store all raw values and weighted values into debug_info array
    debug_info[0] = phase;
    debug_info[1] = material_score;
    debug_info[2] = h_coeffs.material_coeff[phase];
    debug_info[3] = mobility_score;
    debug_info[4] = h_coeffs.mobility_coeff[phase];
    debug_info[5] = corner_score;
    debug_info[6] = h_coeffs.corners_coeff[phase];
    debug_info[7] = parity_score;
    debug_info[8] = h_coeffs.parity_coeff[phase];
    debug_info[9] = stability_score;
    debug_info[10] = h_coeffs.stability_coeff[phase];
    debug_info[11] = frontier_score;
    debug_info[12] = h_coeffs.frontier_coeff[phase];
    debug_info[13] = material_contrib;
    debug_info[14] = mobility_contrib;
    debug_info[15] = corner_contrib;
    debug_info[16] = parity_contrib;
    debug_info[17] = stability_contrib;
    debug_info[18] = frontier_contrib;
    debug_info[19] = final_score;

    // Print full details
    printf("[DEBUG-HOST] P%d Phase=%d: Mat(%d*%d=%d) Mob(%d*%d=%d) Cor(%d*%d=%d) Par(%d*%d=%d) Stb(%d*%d=%d) Frt(%d*%d=%d) => %d\n",
           player_color, phase,
           h_coeffs.material_coeff[phase], material_score, material_contrib,
           h_coeffs.mobility_coeff[phase], mobility_score, mobility_contrib,
           h_coeffs.corners_coeff[phase], corner_score, corner_contrib,
           h_coeffs.parity_coeff[phase], parity_score, parity_contrib,
           h_coeffs.stability_coeff[phase], stability_score, stability_contrib,
           h_coeffs.frontier_coeff[phase], frontier_score, frontier_contrib,
           final_score);
  }

  return final_score;
}
