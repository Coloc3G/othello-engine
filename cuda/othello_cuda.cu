#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "othello_cuda.h"

// Constant board size
#define BOARD_SIZE 8
#define EMPTY 0
#define WHITE 1
#define BLACK 2

// Increase BLOCK_SIZE for better GPU utilization
#define BLOCK_SIZE 256

// Maximum number of positions to store in transposition table
#define MAX_POSITIONS_POOL 65536
#define POSITIONS_BATCH_SIZE 1024
#define MAX_ZOBRIST_ENTRIES 1048576

// Coefficient structure for evaluation
typedef struct
{
  int material_coeff[3];
  int mobility_coeff[3];
  int corners_coeff[3];
  int parity_coeff[3];
  int stability_coeff[3];
  int frontier_coeff[3];
} EvaluationCoefficients;

// Game state structure
typedef struct
{
  int board[BOARD_SIZE][BOARD_SIZE];
  int player_color;
} GameState;

// Transposition table entry
typedef struct
{
  unsigned long long key;
  int score;
  int depth;
  int best_move_row;
  int best_move_col;
} TranspositionEntry;

// Position cache for batching
typedef struct
{
  GameState states[MAX_POSITIONS_POOL];
  int scores[MAX_POSITIONS_POOL];
  unsigned long long hashes[MAX_POSITIONS_POOL];
  int cache_size;
} PositionCache;

// Global device variables
__constant__ EvaluationCoefficients d_coeffs;
__device__ unsigned long long d_zobrist_table[3][BOARD_SIZE][BOARD_SIZE];
__device__ TranspositionEntry d_tt[MAX_ZOBRIST_ENTRIES];
__device__ int d_tt_size = 0;
__device__ PositionCache d_position_cache;

// Host-side copies
EvaluationCoefficients h_coeffs;
unsigned long long h_zobrist_table[3][BOARD_SIZE][BOARD_SIZE];
hipError_t cuda_status = hipSuccess;

//-----------------------------------------------------------------------
// Zobrist hashing utilities
//-----------------------------------------------------------------------

// Compute Zobrist hash for a board position (device)
__device__ unsigned long long computeZobristHash(int board[BOARD_SIZE][BOARD_SIZE], int player)
{
  unsigned long long hash = player; // Include player in hash

  for (int row = 0; row < BOARD_SIZE; row++)
  {
    for (int col = 0; col < BOARD_SIZE; col++)
    {
      if (board[row][col] != EMPTY)
      {
        hash ^= d_zobrist_table[board[row][col] - 1][row][col];
      }
    }
  }

  return hash;
}

// Compute Zobrist hash for a board position (host)
unsigned long long computeZobristHashHost(int board[BOARD_SIZE][BOARD_SIZE], int player)
{
  unsigned long long hash = player; // Include player in hash

  for (int row = 0; row < BOARD_SIZE; row++)
  {
    for (int col = 0; col < BOARD_SIZE; col++)
    {
      if (board[row][col] != EMPTY)
      {
        hash ^= h_zobrist_table[board[row][col] - 1][row][col];
      }
    }
  }

  return hash;
}

//-----------------------------------------------------------------------
// Transposition table utilities
//-----------------------------------------------------------------------

// Store entry in transposition table (device)
__device__ void storeTranspositionEntry(unsigned long long key, int score,
                                        int depth, int best_move_row, int best_move_col)
{
  // Use key as index with modulo to handle collisions
  int index = key % MAX_ZOBRIST_ENTRIES;

  // Always replace for now (could implement more sophisticated replacement policy)
  d_tt[index].key = key;
  d_tt[index].score = score;
  d_tt[index].depth = depth;
  d_tt[index].best_move_row = best_move_row;
  d_tt[index].best_move_col = best_move_col;

  // Atomic increment to track table size
  atomicMin(&d_tt_size, MAX_ZOBRIST_ENTRIES);
}

// Lookup entry in transposition table (device)
__device__ bool lookupTranspositionEntry(unsigned long long key, int depth,
                                         int *score, int *best_move_row, int *best_move_col)
{
  int index = key % MAX_ZOBRIST_ENTRIES;

  // Check if we have a valid entry with sufficient depth
  if (d_tt[index].key == key && d_tt[index].depth >= depth)
  {
    *score = d_tt[index].score;
    *best_move_row = d_tt[index].best_move_row;
    *best_move_col = d_tt[index].best_move_col;
    return true;
  }

  return false;
}

//-----------------------------------------------------------------------
// Device-only functions (run on GPU)
//-----------------------------------------------------------------------

// Check if a move is valid for the given board and player
__device__ bool isValidMove(int board[BOARD_SIZE][BOARD_SIZE], int player, int row, int col)
{
  // Check if the position is empty
  if (board[row][col] != EMPTY)
    return false;

  // Get opponent color
  int opponent = (player == WHITE) ? BLACK : WHITE;

  // Direction vectors for all 8 directions
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  // Check all 8 directions
  for (int dir = 0; dir < 8; dir++)
  {
    int r = row + dx[dir];
    int c = col + dy[dir];

    // First step must have opponent piece
    if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE && board[r][c] == opponent)
    {
      // Continue in this direction
      r += dx[dir];
      c += dy[dir];

      // Keep going until we find an empty cell, edge, or our own piece
      while (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE)
      {
        if (board[r][c] == EMPTY)
          break;
        if (board[r][c] == player)
          return true; // Found our own piece, move is valid

        // Continue in this direction
        r += dx[dir];
        c += dy[dir];
      }
    }
  }

  return false;
}

// Apply a move to the board and return a new board
__device__ void applyMove(int original[BOARD_SIZE][BOARD_SIZE], int result[BOARD_SIZE][BOARD_SIZE],
                          int player, int row, int col)
{
  // Copy the original board
  for (int i = 0; i < BOARD_SIZE; i++)
    for (int j = 0; j < BOARD_SIZE; j++)
      result[i][j] = original[i][j];

  // Place the piece
  result[row][col] = player;

  // Get opponent color
  int opponent = (player == WHITE) ? BLACK : WHITE;

  // Direction vectors for all 8 directions
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  // Check all 8 directions and flip pieces
  for (int dir = 0; dir < 8; dir++)
  {
    int r = row + dx[dir];
    int c = col + dy[dir];

    // Pieces to flip in this direction
    int flip_r[BOARD_SIZE * BOARD_SIZE], flip_c[BOARD_SIZE * BOARD_SIZE];
    int flip_count = 0;

    // Check if first piece is opponent
    if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE && result[r][c] == opponent)
    {
      // Remember this piece
      flip_r[flip_count] = r;
      flip_c[flip_count] = c;
      flip_count++;

      // Continue in this direction
      r += dx[dir];
      c += dy[dir];

      // Find all opponent pieces in this direction
      while (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE)
      {
        if (result[r][c] == EMPTY)
        {
          flip_count = 0; // No pieces to flip
          break;
        }

        if (result[r][c] == player)
          break; // Found our piece, can flip

        // Remember opponent piece
        flip_r[flip_count] = r;
        flip_c[flip_count] = c;
        flip_count++;

        // Continue in this direction
        r += dx[dir];
        c += dy[dir];
      }

      // If we found our piece at the end, flip all pieces in between
      if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE && result[r][c] == player)
      {
        for (int i = 0; i < flip_count; i++)
          result[flip_r[i]][flip_c[i]] = player;
      }
    }
  }
}

// Get all valid moves for a player
__device__ int getValidMoves(int board[BOARD_SIZE][BOARD_SIZE], int player, int moves_r[64], int moves_c[64])
{
  int count = 0;
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (isValidMove(board, player, i, j))
      {
        moves_r[count] = i;
        moves_c[count] = j;
        count++;
      }
    }
  }
  return count;
}

// Evaluate stability of pieces
__device__ int evaluateStability(int board[BOARD_SIZE][BOARD_SIZE], int player, int opponent)
{
  // Pre-computed stability weights
  const int stability_map[BOARD_SIZE][BOARD_SIZE] = {
      {4, -3, 2, 2, 2, 2, -3, 4},
      {-3, -4, -1, -1, -1, -1, -4, -3},
      {2, -1, 1, 0, 0, 1, -1, 2},
      {2, -1, 0, 1, 1, 0, -1, 2},
      {2, -1, 0, 1, 1, 0, -1, 2},
      {2, -1, 1, 0, 0, 1, -1, 2},
      {-3, -4, -1, -1, -1, -1, -4, -3},
      {4, -3, 2, 2, 2, 2, -3, 4}};

  int player_stability = 0;
  int opponent_stability = 0;

  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board[i][j] == player)
      {
        player_stability += stability_map[i][j];
      }
      else if (board[i][j] == opponent)
      {
        opponent_stability += stability_map[i][j];
      }
    }
  }

  return player_stability - opponent_stability;
}

// Count frontier discs (adjacent to empty spaces)
__device__ int countFrontierDiscs(int board[BOARD_SIZE][BOARD_SIZE], int player)
{
  int count = 0;
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board[i][j] == player)
      {
        // Check if this piece is adjacent to an empty square
        for (int dir = 0; dir < 8; dir++)
        {
          int r = i + dx[dir];
          int c = j + dy[dir];

          if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE &&
              board[r][c] == EMPTY)
          {
            count++;
            break; // Count each piece only once
          }
        }
      }
    }
  }

  return count;
}

// Enhanced board evaluation with more heuristics
__device__ int evaluateBoard(int board[BOARD_SIZE][BOARD_SIZE], int player, int phase)
{
  int opponent = (player == WHITE) ? BLACK : WHITE;

  // Material evaluation
  int player_pieces = 0;
  int opponent_pieces = 0;
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board[i][j] == player)
        player_pieces++;
      else if (board[i][j] == opponent)
        opponent_pieces++;
    }
  }
  int material_score = player_pieces - opponent_pieces;

  // Corner evaluation
  int player_corners = 0;
  int opponent_corners = 0;
  if (board[0][0] == player)
    player_corners++;
  if (board[0][7] == player)
    player_corners++;
  if (board[7][0] == player)
    player_corners++;
  if (board[7][7] == player)
    player_corners++;

  if (board[0][0] == opponent)
    opponent_corners++;
  if (board[0][7] == opponent)
    opponent_corners++;
  if (board[7][0] == opponent)
    opponent_corners++;
  if (board[7][7] == opponent)
    opponent_corners++;

  int corner_score = 100 * (player_corners - opponent_corners) / (player_corners + opponent_corners + 1);

  // Mobility evaluation
  int moves_r[64], moves_c[64];
  int player_moves = getValidMoves(board, player, moves_r, moves_c);
  int opponent_moves = getValidMoves(board, opponent, moves_r, moves_c);
  int mobility_score = 100 * (player_moves - opponent_moves) / (player_moves + opponent_moves + 1);

  // Parity evaluation (beneficial to have the last move)
  int empty_squares = 64 - player_pieces - opponent_pieces;
  int parity_score = (empty_squares % 2 == 0) ? -1 : 1; // Even is bad for next player

  // Stability evaluation
  int stability_score = evaluateStability(board, player, opponent);

  // Frontier evaluation
  int player_frontier = countFrontierDiscs(board, player);
  int opponent_frontier = countFrontierDiscs(board, opponent);
  int frontier_score = -100 * (player_frontier - opponent_frontier) / (player_frontier + opponent_frontier + 1);

  // Final score using coefficients
  return d_coeffs.material_coeff[phase] * material_score +
         d_coeffs.mobility_coeff[phase] * mobility_score +
         d_coeffs.corners_coeff[phase] * corner_score +
         d_coeffs.parity_coeff[phase] * parity_score +
         d_coeffs.stability_coeff[phase] * stability_score +
         d_coeffs.frontier_coeff[phase] * frontier_score;
}

// CUDA kernel to evaluate multiple game states in parallel with shared memory
__global__ void evaluateStatesKernel(GameState *states, int *scores, int num_states)
{
  // Use shared memory for faster access
  __shared__ int shared_results[BLOCK_SIZE];

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;

  if (idx < num_states)
  {
    GameState state = states[idx];
    int board[BOARD_SIZE][BOARD_SIZE];
    int player_color = state.player_color;

    // Copy the board to local memory for faster access
    for (int i = 0; i < BOARD_SIZE; i++)
    {
      for (int j = 0; j < BOARD_SIZE; j++)
      {
        board[i][j] = state.board[i][j];
      }
    }

    // Determine game phase
    int piece_count = 0;
    for (int i = 0; i < BOARD_SIZE; i++)
    {
      for (int j = 0; j < BOARD_SIZE; j++)
      {
        if (board[i][j] != EMPTY)
        {
          piece_count++;
        }
      }
    }

    int phase;
    if (piece_count < 20)
      phase = 0;
    else if (piece_count <= 58)
      phase = 1;
    else
      phase = 2;

    // Calculate and store the evaluation score
    shared_results[tid] = evaluateBoard(board, player_color, phase);
  }
  else
  {
    // Default value for unused threads
    shared_results[tid] = 0;
  }

  // Synchronize threads in the block
  __syncthreads();

  // Copy result to global memory
  if (idx < num_states)
  {
    scores[idx] = shared_results[tid];
  }
}

//-----------------------------------------------------------------------
// Host-only code (CPU side)
//-----------------------------------------------------------------------

// Host function to check if a move is valid (CPU implementation)
bool isValidMoveHost(int board[BOARD_SIZE][BOARD_SIZE], int player, int row, int col)
{
  if (board[row][col] != EMPTY)
    return false;

  int opponent = (player == WHITE) ? BLACK : WHITE;
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  for (int dir = 0; dir < 8; dir++)
  {
    int r = row + dx[dir];
    int c = col + dy[dir];

    if (r < 0 || r >= BOARD_SIZE || c < 0 || c >= BOARD_SIZE || board[r][c] != opponent)
      continue;

    r += dx[dir];
    c += dy[dir];
    bool foundPlayerPiece = false;

    while (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE)
    {
      if (board[r][c] == EMPTY)
        break;
      if (board[r][c] == player)
      {
        foundPlayerPiece = true;
        break;
      }
      r += dx[dir];
      c += dy[dir];
    }

    if (foundPlayerPiece)
      return true;
  }

  return false;
}

// Host function to apply a move (CPU implementation)
void applyMoveHost(int board[BOARD_SIZE][BOARD_SIZE], int player, int row, int col)
{
  board[row][col] = player;
  int opponent = (player == WHITE) ? BLACK : WHITE;
  int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};
  int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};

  for (int dir = 0; dir < 8; dir++)
  {
    int r = row + dx[dir];
    int c = col + dy[dir];

    if (r < 0 || r >= BOARD_SIZE || c < 0 || c >= BOARD_SIZE || board[r][c] != opponent)
      continue;

    // Store positions to flip
    int flipPositions[BOARD_SIZE * BOARD_SIZE][2];
    int flipCount = 0;

    while (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE && board[r][c] == opponent)
    {
      flipPositions[flipCount][0] = r;
      flipPositions[flipCount][1] = c;
      flipCount++;
      r += dx[dir];
      c += dy[dir];
    }

    // If we found our piece at the end, flip all pieces in between
    if (r >= 0 && r < BOARD_SIZE && c >= 0 && c < BOARD_SIZE && board[r][c] == player)
    {
      for (int i = 0; i < flipCount; i++)
      {
        board[flipPositions[i][0]][flipPositions[i][1]] = player;
      }
    }
  }
}

// Host function to get all valid moves (CPU implementation)
int getValidMovesHost(int board[BOARD_SIZE][BOARD_SIZE], int player, int moves_r[64], int moves_c[64])
{
  int count = 0;
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (isValidMoveHost(board, player, i, j))
      {
        moves_r[count] = i;
        moves_c[count] = j;
        count++;
      }
    }
  }
  return count;
}

// Evaluate a board state (CPU implementation)
int evaluateBoardHost(int board[BOARD_SIZE][BOARD_SIZE], int player, EvaluationCoefficients coeffs)
{
  int opponent = (player == WHITE) ? BLACK : WHITE;

  // Count pieces for phase determination
  int piece_count = 0;
  int player_pieces = 0;
  int opponent_pieces = 0;

  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (board[i][j] != EMPTY)
      {
        piece_count++;
        if (board[i][j] == player)
          player_pieces++;
        else if (board[i][j] == opponent)
          opponent_pieces++;
      }
    }
  }

  int phase;
  if (piece_count < 20)
    phase = 0; // Early game
  else if (piece_count <= 58)
    phase = 1; // Mid game
  else
    phase = 2; // Late game

  int material_score = player_pieces - opponent_pieces;

  // Corner evaluation
  int player_corners = 0;
  int opponent_corners = 0;
  if (board[0][0] == player)
    player_corners++;
  if (board[0][7] == player)
    player_corners++;
  if (board[7][0] == player)
    player_corners++;
  if (board[7][7] == player)
    player_corners++;

  if (board[0][0] == opponent)
    opponent_corners++;
  if (board[0][7] == opponent)
    opponent_corners++;
  if (board[7][0] == opponent)
    opponent_corners++;
  if (board[7][7] == opponent)
    opponent_corners++;

  int corner_score = 100 * (player_corners - opponent_corners) / (player_corners + opponent_corners + 1);

  // Mobility calculation
  int moves_r[64], moves_c[64];
  int player_moves = getValidMovesHost(board, player, moves_r, moves_c);
  int opponent_moves = getValidMovesHost(board, opponent, moves_r, moves_c);
  int mobility_score = 100 * (player_moves - opponent_moves) / (player_moves + opponent_moves + 1);

  return coeffs.material_coeff[phase] * material_score +
         coeffs.corners_coeff[phase] * corner_score +
         coeffs.mobility_coeff[phase] * mobility_score;
}

// Recursive minimax search (CPU implementation)
int minimaxHost(int board[BOARD_SIZE][BOARD_SIZE], int player, int depth, bool maximizing,
                int alpha, int beta, int *best_row, int *best_col, EvaluationCoefficients coeffs)
{
  // Leaf node evaluation
  if (depth == 0)
  {
    return evaluateBoardHost(board, maximizing ? player : (player == WHITE ? BLACK : WHITE), coeffs);
  }

  int opponent = (player == WHITE) ? BLACK : WHITE;
  int moves_r[64], moves_c[64];
  int move_count = getValidMovesHost(board, player, moves_r, moves_c);

  // No moves - check if game is over or we need to pass
  if (move_count == 0)
  {
    int opp_moves_r[64], opp_moves_c[64];
    int opp_move_count = getValidMovesHost(board, opponent, opp_moves_r, opp_moves_c);

    // Game is over
    if (opp_move_count == 0)
    {
      return evaluateBoardHost(board, maximizing ? player : opponent, coeffs);
    }

    // Pass turn
    return minimaxHost(board, opponent, depth - 1, !maximizing, alpha, beta, best_row, best_col, coeffs);
  }

  // Maximizing player
  if (maximizing)
  {
    int best_score = -1000000;
    int best_move_r = -1;
    int best_move_c = -1;

    for (int i = 0; i < move_count; i++)
    {
      // Copy board
      int new_board[BOARD_SIZE][BOARD_SIZE];
      for (int r = 0; r < BOARD_SIZE; r++)
        for (int c = 0; c < BOARD_SIZE; c++)
          new_board[r][c] = board[r][c];

      // Apply move
      applyMoveHost(new_board, player, moves_r[i], moves_c[i]);

      // Recursive search
      int dummy_r = -1, dummy_c = -1;
      int score = minimaxHost(new_board, opponent, depth - 1, false, alpha, beta, &dummy_r, &dummy_c, coeffs);

      if (score > best_score)
      {
        best_score = score;
        best_move_r = moves_r[i];
        best_move_c = moves_c[i];
      }

      // Alpha-beta pruning
      alpha = alpha > best_score ? alpha : best_score;
      if (beta <= alpha)
        break;
    }

    *best_row = best_move_r;
    *best_col = best_move_c;
    return best_score;
  }
  // Minimizing player
  else
  {
    int best_score = 1000000;
    int best_move_r = -1;
    int best_move_c = -1;

    for (int i = 0; i < move_count; i++)
    {
      // Copy board
      int new_board[BOARD_SIZE][BOARD_SIZE];
      for (int r = 0; r < BOARD_SIZE; r++)
        for (int c = 0; c < BOARD_SIZE; c++)
          new_board[r][c] = board[r][c];

      // Apply move
      applyMoveHost(new_board, player, moves_r[i], moves_c[i]);

      // Recursive search
      int dummy_r = -1, dummy_c = -1;
      int score = minimaxHost(new_board, opponent, depth - 1, true, alpha, beta, &dummy_r, &dummy_c, coeffs);

      if (score < best_score)
      {
        best_score = score;
        best_move_r = moves_r[i];
        best_move_c = moves_c[i];
      }

      // Alpha-beta pruning
      beta = beta < best_score ? beta : best_score;
      if (beta <= alpha)
        break;
    }

    *best_row = best_move_r;
    *best_col = best_move_c;
    return best_score;
  }
}

//-----------------------------------------------------------------------
// External C interface (exported functions)
//-----------------------------------------------------------------------

// Initialize CUDA and return success status
__declspec(dllexport) int initCUDA()
{
  hipError_t error;
  int deviceCount;

  error = hipGetDeviceCount(&deviceCount);
  if (error != hipSuccess)
  {
    printf("CUDA Error: %s\n", hipGetErrorString(error));
    return 0;
  }

  if (deviceCount == 0)
  {
    printf("No CUDA-capable devices found\n");
    return 0;
  }

  // Choose device 0 by default
  error = hipSetDevice(0);
  if (error != hipSuccess)
  {
    printf("CUDA Error: %s\n", hipGetErrorString(error));
    return 0;
  }

  return 1;
}

// Initialize Zobrist hash table
__declspec(dllexport) void initZobristTable()
{
  srand((unsigned int)time(NULL));
  for (int piece = 0; piece < 3; piece++)
  {
    for (int row = 0; row < BOARD_SIZE; row++)
    {
      for (int col = 0; col < BOARD_SIZE; col++)
      {
        // Generate random 64-bit value for each board position and piece
        h_zobrist_table[piece][row][col] =
            ((unsigned long long)rand() << 32) | rand();
      }
    }
  }

  // Copy Zobrist table to device
  hipMemcpyToSymbol(HIP_SYMBOL(d_zobrist_table), h_zobrist_table,
                     sizeof(h_zobrist_table));
}

// Set evaluation coefficients
__declspec(dllexport) void setCoefficients(int *material, int *mobility, int *corners,
                                           int *parity, int *stability, int *frontier)
{
  EvaluationCoefficients h_coeffs;

  // Copy coefficients from host arrays to host struct
  for (int i = 0; i < 3; i++)
  {
    h_coeffs.material_coeff[i] = material[i];
    h_coeffs.mobility_coeff[i] = mobility[i];
    h_coeffs.corners_coeff[i] = corners[i];
    h_coeffs.parity_coeff[i] = parity[i];
    h_coeffs.stability_coeff[i] = stability[i];
    h_coeffs.frontier_coeff[i] = frontier[i];
  }

  // Copy coefficients to device constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(d_coeffs), &h_coeffs, sizeof(EvaluationCoefficients));
}

// Evaluate multiple game states in parallel
__declspec(dllexport) void evaluateStates(int *boards, int *player_colors, int *scores, int num_states)
{
  // Measure transfer and execution time for profiling
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  GameState *h_states = (GameState *)malloc(num_states * sizeof(GameState));
  GameState *d_states;
  int *d_scores;

  // Prepare game states
  for (int s = 0; s < num_states; s++)
  {
    h_states[s].player_color = player_colors[s];
    for (int i = 0; i < BOARD_SIZE; i++)
    {
      for (int j = 0; j < BOARD_SIZE; j++)
      {
        h_states[s].board[i][j] = boards[s * BOARD_SIZE * BOARD_SIZE + i * BOARD_SIZE + j];
      }
    }
  }

  // Allocate device memory and copy data - use pinned memory for faster transfers
  hipMalloc((void **)&d_states, num_states * sizeof(GameState));
  hipMalloc((void **)&d_scores, num_states * sizeof(int));

  // Copy data to device
  hipMemcpy(d_states, h_states, num_states * sizeof(GameState), hipMemcpyHostToDevice);

  // Calculate grid dimensions
  int threads = BLOCK_SIZE;
  int blocks = (num_states + threads - 1) / threads;

  // Launch kernel
  evaluateStatesKernel<<<blocks, threads>>>(d_states, d_scores, num_states);

  // Check for kernel launch errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("CUDA Kernel Error: %s\n", hipGetErrorString(err));
    // In case of error, zero all scores
    memset(scores, 0, num_states * sizeof(int));

    // Free memory and return
    hipFree(d_states);
    hipFree(d_scores);
    free(h_states);
    return;
  }

  // Copy results back to host
  hipMemcpy(scores, d_scores, num_states * sizeof(int), hipMemcpyDeviceToHost);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  if (num_states > 1000)
  {
    printf("GPU processed %d states in %.2f ms (%.2f states/ms)\n",
           num_states, milliseconds, num_states / milliseconds);
  }

  // Free memory
  hipFree(d_states);
  hipFree(d_scores);
  free(h_states);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

// Find the best move using minimax
__declspec(dllexport) int findBestMove(int *board, int player_color, int depth, int *best_row, int *best_col)
{
  // Convert the flat board array to 2D array
  int board_2d[BOARD_SIZE][BOARD_SIZE];
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      board_2d[i][j] = board[i * BOARD_SIZE + j];
    }
  }

  // Get current evaluation coefficients from device
  EvaluationCoefficients h_coeffs;
  hipMemcpyFromSymbol(&h_coeffs, HIP_SYMBOL(d_coeffs), sizeof(EvaluationCoefficients));

  // Initialize best move coordinates
  int br = -1, bc = -1;

  // Perform minimax search
  int best_score = minimaxHost(
      board_2d,     // game board
      player_color, // current player
      depth,        // search depth
      true,         // maximizing player
      -1000000,     // alpha
      1000000,      // beta
      &br,          // best row (output)
      &bc,          // best column (output)
      h_coeffs      // evaluation coefficients
  );

  // Set output parameters
  *best_row = br;
  *best_col = bc;

  return best_score;
}

// Check if a player has valid moves
__declspec(dllexport) int hasValidMoves(int *board, int player_color)
{
  // Convert the flat board array to 2D array
  int board_2d[BOARD_SIZE][BOARD_SIZE];
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      board_2d[i][j] = board[i * BOARD_SIZE + j];
    }
  }

  // Check for any valid move
  for (int i = 0; i < BOARD_SIZE; i++)
  {
    for (int j = 0; j < BOARD_SIZE; j++)
    {
      if (isValidMoveHost(board_2d, player_color, i, j))
      {
        return 1; // At least one valid move exists
      }
    }
  }

  return 0; // No valid moves
}

// Check if game is finished (no valid moves for either player)
__declspec(dllexport) int isGameFinished(int *board)
{
  // No player can move = game is finished
  return !hasValidMoves(board, BLACK) && !hasValidMoves(board, WHITE);
}

// Get GPU memory information
__declspec(dllexport) void getGPUMemoryInfo(unsigned long long *free_memory, unsigned long long *total_memory)
{
  size_t free, total;
  hipMemGetInfo(&free, &total);

  *free_memory = free;
  *total_memory = total;
}

// Free CUDA resources
__declspec(dllexport) void cleanupCUDA()
{
  hipDeviceReset();
}
